#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>

using namespace std;

#define N 300000
#define M 30

__global__ void vectorAdd(float *a, float *b, float *c, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

__global__ void matrixMul(float *A, float *B, float *C, int m)
{
    int row = threadIdx.y + bloclIdx.y * blockDim * y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < m && col < m)
    {
        float sum = 0;
        for (int k = 0; k < m; ++k)
        {
            sum += A[row * m + k] * B[k * m + col];
        }
        C[row * m + col] = sum;
    }
}

int main()
{
    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
    h_a = new float[N];
    h_b = new float[N];
    h_c = new float[N];

    vector<float> cpu_c_vector(N);

    for (int i = 0; i < N; i++)
    {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    auto start_cpu = chrono::high_resolution_clock::now();
    for (int i = 0; i < N; i++)
    {
        cpu_c_vector[i] = h_a[i] + h_b[i];
    }

    auto end_cpu = chrono::high_resolution_clock::now();
    double cpu_time = chrono::duration<double>(end_cpu - start_cpu).count();

    hipEvent_t start, stop;
    float gpu_time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start);
    vectorAdd<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);

    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_time, start, stop);
    cout << "CPU TIme " << cpu_time << " sec\n";
    cout << "GPU TIme " << gpu_time << " sec\n";
    cout << "SpeedUp: " << cpu_time / (gpu_time / 1000.0) << "\n";

    bool vector_match = true;

    for (int i = 0; i < N; i++)
    {
        if (fabs(h_c[i] - cpu_c_vector[i]) > 1e-5)
        {
            vector_match = false;
            break;
        }
    }

    cout << "Vector Addtion Result " << (vector_match ? "matched" : "not matched") << "\n\n";

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[M * M];
    h_B = new float[M * M];
    h_C = new float[M * M];

    vector<float> cpu_C_vector(M * M);

    for (int i = 0; i < M * M; i++)
    {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    hipMalloc(&d_A, M * M * sizeof(float));
    hipMalloc(&d_B, M * M * sizeof(float));
    hipMalloc(&d_C, M * M * sizeof(float));

    start_cpu = chrono::high_resolution_clock::now();
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < M; j++)
        {
            float sum = 0;
            for (int k = 0; k < M; k++)
            {
                sum += h_A[i * M + k] + h_B[k * M + j];
            }
            cpu_C_vector[i * M + j] = sum;
        }
    }
    end_cpu = chrono::high_resolution_clock::now();
    cpu_time = chrono::duration<double>(end_cpu - start_cpu).count();

    hipMemcpy(d_A, h_A, M * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, M * M * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                    (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEventRecord(start);
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M);
    hipEventRecord(stop);

    hipMemcpy(h_C, d_C, M * M * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    cout << "CPU TIme " << cpu_time << "sec\n\n";
    cout << "GPU TIme " << gpu_time / 1000.0 << "sec\n\n";
    cout << "SpeedUp " << cpu_time / (gpu_time / 1000.0) << "\n\n";

    bool matrix_match = true;

    for (int i = 0; i < M*M; i++)
    {
        if (fabs(h_C[i] - cpu_C_vector[i]) > 1e-5)
        {
            matrix_match = false;
            break;
        }
    }

    cout << "Matrix Result " << (matrix_match ? "Match" : "Not Matched");

    delete[] h_a, h_b, h_c, h_A, h_B, h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}